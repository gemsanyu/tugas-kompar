#include "hip/hip_runtime.h"
#include<stdio.h>
#include<random>
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void mergeSort(int *array, int *left, int *right){
  if (*left == *right){
    return;
  }

  if (*left == *right-1){
    if (array[*left]>array[*right]){
      int c = array[*left];
      array[*left]=array[*right];
      array[*right]=c;
    }
    return;
  }

  printf("%d %d\n", *left, *right);
  hipStream_t s1,s2;

  int mid = ((*left)+(*right))/2;
  int *a=left;

  int *b=new int(mid);
  int *a1=new int(mid+1);
  int *b1=right;

  int *arr, *arr1;
  hipMalloc(&arr, (*b-*a+1)*sizeof(int));
  memcpy(arr, array, (*b-*a+1)*sizeof(int));
  hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
  mergeSort<<< 1,1,0,s1 >>>(arr, a, b);

  hipMalloc(&arr1, (*b1-*a1+1)*sizeof(int));
  memcpy(arr1, array, (*b1-*a1+1)*sizeof(int));
  hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
  mergeSort<<< 1,1,0,s2 >>>(arr1, a1, b1);

  hipStreamSynchronize(s1);
  int c=*a;
  int c1=*a1;
  for(int idx=*left; idx<=*right; idx++){
    if((c1>*b1) || arr[c]<=arr1[c1]){
      array[idx]=arr[c];
      c++;
    } else {
      array[idx]=arr1[c1];
      c1++;
    }
  }

  for(int i=*left;i<=*right;i++){
    printf("%d-",array[i]);
  }
  printf("\n");

  hipFree(arr);
  hipFree(arr1);
  hipFree(a);
  hipFree(b);
  hipFree(a1);
  hipFree(b1);
  hipStreamDestroy(s1);
  hipStreamDestroy(s2);
}

__global__ void invokeMergeSort(int *array, int *left, int *right){
  mergeSort<<<1,1>>>(array, left, right);
}

int main(int argc, char** argv){
  int N = atoi(argv[1])-1;
  int *d_N;
  hipMalloc(&d_N, sizeof(int));
  hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice);
  N++;
  int *h_arr, *d_arr;
  h_arr = (int*)malloc(N*sizeof(int));
  for(int i=0;i<N;i++){
    h_arr[i]=rand()%1000;
  }
  for(int i=0;i<N;i++){
    printf("%d-",h_arr[i]);
  }
  printf("\n");

  hipMalloc(&d_arr, N*sizeof(int));
  hipMemcpy(d_arr, h_arr, N*sizeof(int), hipMemcpyHostToDevice);

  int left=0;
  int *d_left;
  hipMalloc(&d_left, sizeof(int));
  hipMemcpy(d_left, &left, sizeof(int), hipMemcpyHostToDevice);

  invokeMergeSort<<< 1,1,0,0 >>>(d_arr, d_left, d_N);
  hipDeviceSynchronize();
  hipMemcpy(h_arr, d_arr, N*sizeof(int), hipMemcpyDeviceToHost);
  // for(int i=0;i<N;i++){
  //   printf("%d-",h_arr[i]);
  // }
  // printf("\n");
}
